#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>


using namespace cv;
using namespace std;
__global__ void rgba_to_greyscale(const uchar4* const rgbaImage,
	unsigned char* greyImage,
	int numRows, int numCols)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	__syncthreads();
	if (col >= numCols || row >= numRows) {
		return;
	}

	int offset = row * numCols + col;

	uchar4 rgba_pixel = rgbaImage[offset];
	float greyness = .299f * rgba_pixel.x + .587f * rgba_pixel.y +
		.114f * rgba_pixel.z;
	greyImage[offset] = static_cast<unsigned char>(greyness);
}

int main()
{
	Mat imageRGBA;
	Mat imageGrey;
	uchar4        *h_rgbaImage;
	uchar4 *d_rgbaImage = NULL;
	unsigned char *h_greyImage;
	unsigned char *d_greyImage = NULL;
	///////////////////////////////////
	Mat image;
	image = cv::imread("IMG.jpg");
	if (image.empty()) {
		cerr << "Couldn't open file: " << endl;
		exit(1);
	}
	imshow("input", image);
	waitKey(0);
	destroyAllWindows();
	///////////////////////////////////
	int numRows = image.rows;
	int numCols = image.cols;
	///////////////////////////////////////
	cvtColor(image, imageRGBA, COLOR_BGR2RGBA);

	//Allocate Memory for output
	imageGrey.create(image.rows, image.cols, CV_8UC1);

	//h_rgbaImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
	//h_greyImage = imageGrey.ptr<unsigned char>(0);
	h_rgbaImage = (uchar4 *)imageRGBA.data;
	h_greyImage = (unsigned char *)imageGrey.data;

	const size_t numPixels = numRows * numCols;

	//Allocate memory on the device for both input and output

	hipMalloc((void**)&d_rgbaImage, sizeof(uchar4) * numPixels);
	hipMalloc((void**)&d_greyImage, sizeof(unsigned char) * numPixels);
	hipMemset((void *)d_greyImage, 0, numPixels * sizeof(unsigned char));
	//Copy input array to the GPU

	hipMemcpy(d_rgbaImage, h_rgbaImage, sizeof(uchar4)*numPixels, hipMemcpyHostToDevice);

	//Calling the Kernel - 

	const dim3 blockSize(32, 16, 1);
	const dim3 gridSize(1 + (numCols / blockSize.x), 1 + (numRows / blockSize.y), 1);

	rgba_to_greyscale <<<gridSize, blockSize >>> (d_rgbaImage, d_greyImage, numRows, numCols);

	//Copy Output array to Host

	hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost);

	//Check Output
	Mat output;
	output = Mat(numRows, numCols, CV_8UC1, (void*)h_greyImage);
	imwrite("result.jpg", output);
	
}

