#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace cv;
using namespace std;
__global__
void gaussian_blur(const unsigned char* const inputChannel,
	unsigned char* const outputChannel,
	int numRows, int numCols,
	const float* const filter, const int filterWidth)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//check if the indices are out of bound
	if (col >= numCols || row >= numRows)
	{
		return;
	}
	int index = row * numCols + col;
	float pixelvalue = 0.0f;
	int filter_index = 0;
	for (int i = row - (filterWidth - 1) / 2; i <= row + (filterWidth - 1) / 2; ++i)
	{
		for (int j = row - (filterWidth - 1) / 2; j <= row + (filterWidth - 1) / 2; ++j)
		{
			if (i >= 0 && j >= 0 && i < numRows && j < numCols)
			{
				int id = i * numCols + col;
				pixelvalue += (float)inputChannel[id] * filter[filter_index];
			}
			++filter_index;
		}
	}
	outputChannel[index] = (unsigned char)pixelvalue;
}
__global__
void separateChannels(const uchar4* const inputImageRGBA,
	int numRows,
	int numCols,
	unsigned char* const redChannel,
	unsigned char* const greenChannel,
	unsigned char* const blueChannel)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//check if the indices are out of bound
	if (col >= numCols || row >= numRows)
	{
		return;
	}
	int index = row * numCols + col;
	uchar4 pixel_value = inputImageRGBA[index];
	redChannel[index] = pixel_value.x;
	greenChannel[index] = pixel_value.y;
	blueChannel[index] = pixel_value.z;
}

__global__
void combineChannels(uchar4* const outputImageRGBA,
	int numRows,
	int numCols,
	const unsigned char* const redChannel,
	const unsigned char* const greenChannel,
	const unsigned char* const blueChannel)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//check if the indices are out of bound
	if (col >= numCols || row >= numRows)
	{
		return;
	}
	int index = row * numCols + col;
	uchar4 pixel;
	pixel.x = redChannel[index];
	pixel.y = greenChannel[index];
	pixel.z = blueChannel[index];
	pixel.w = 255;
	outputImageRGBA[index] = pixel;
}

int main()
{
	uchar4 *h_rgbaImage, *d_rgbaImage = NULL;
	unsigned char *h_red, *h_green, *h_blue, *d_red, *d_green, *d_blue = NULL;
	unsigned char *db_red, *db_green, *db_blue = NULL;

	// Read Image
	Mat image;
	image = imread("IMG.jpg");
	if (image.empty()) {
		cerr << "Couldn't open file: " << endl;
		exit(1);
	}
	int numRows = image.rows;
	int numCols = image.cols;
	int numPixels = numCols * numRows;

	Mat imageRGBA;
	cvtColor(image, imageRGBA, COLOR_BGR2RGBA);
	//
	h_rgbaImage = (uchar4 *)imageRGBA.data;
	Mat temp;
	temp.create(image.rows, image.cols, CV_8UC1);
	h_red = (unsigned char *)temp.data;
	h_green = (unsigned char *)temp.data;
	h_blue = (unsigned char *)temp.data;
	// Allocate memory
	hipMalloc((void**)&d_rgbaImage, sizeof(uchar4) * numPixels);
	hipMalloc((void**)&d_red, sizeof(unsigned char)*numPixels);
	hipMalloc((void**)&d_green, sizeof(unsigned char)*numPixels);
	hipMalloc((void**)&d_blue, sizeof(unsigned char)*numPixels);

	hipMemcpy(d_rgbaImage, h_rgbaImage, sizeof(uchar4)*numPixels, hipMemcpyHostToDevice);

	//Calling the Kernel - 

	const dim3 blockSize(32, 16, 1);
	const dim3 gridSize(1 + (numCols / blockSize.x), 1 + (numRows / blockSize.y), 1);

	separateChannels << < gridSize, blockSize >> > (d_rgbaImage, numRows, numCols, d_red, d_green, d_blue);

	float kernel[9] = { float(1) / 16, float(2) / 16, float(1) / 16, float(2) / 16, float(4) / 16, float(2) / 16, float(1) / 16, float(2) / 16, float(1) / 16 };
	

	hipMalloc((void**)&db_red, sizeof(unsigned char)*numPixels);
	hipMalloc((void**)&db_green, sizeof(unsigned char)*numPixels);
	hipMalloc((void**)&db_blue, sizeof(unsigned char)*numPixels);

	float *d_kernel;
	hipMalloc((void**)&d_kernel, sizeof(float) * 9);
	hipMemcpy(d_kernel, kernel, sizeof(float) * 9, hipMemcpyHostToDevice);
	for (int i = 0; i < 9; ++i)
	{
		cout << kernel[i] << '\n';
	}

	gaussian_blur <<< gridSize, blockSize >>> (d_red, db_red, numRows, numCols, d_kernel, 3);
	gaussian_blur <<< gridSize, blockSize >>> (d_green, db_green, numRows, numCols, d_kernel, 3);
	gaussian_blur <<< gridSize, blockSize >>> (d_blue, db_blue, numRows, numCols, d_kernel, 3);
	
	uchar4 *h_outputImage = NULL;
	uchar4 *d_outputImage = NULL;
	Mat temp2;
	cvtColor(temp, temp2, COLOR_BGR2RGBA);
	h_outputImage = (uchar4 *)temp2.data;

	hipMalloc((void**)&d_outputImage, sizeof(uchar4) * numPixels);

	combineChannels << <gridSize, blockSize >> > (d_outputImage, numRows, numCols, db_red, db_green, db_blue);

	hipMemcpy(h_outputImage, d_outputImage, sizeof(uchar4)*numPixels, hipMemcpyDeviceToHost);

	Mat output;
	output = Mat(numRows, numCols, CV_8UC4, (void*)h_outputImage);
	
	Mat imageOutputBGR;
	cvtColor(output, imageOutputBGR, COLOR_RGBA2BGR);
	imwrite("result.jpg", imageOutputBGR);
	
}
